// Assignment For Module 03:
// Blocks, Warps and Threads

// Author: Justin Renga


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/// @brief The Kernel function that will execute on the GPU.
///
/// @param [inout] input1    The first input array (contains integers)
/// @param [inout] input2    The second input array (contains integers)
/// @param [inout] output    The resulting array (first + second)
__global__ void integer_addition(int* input1, int* input2, int* output)
{
  const unsigned int thread_index = (blockIdx.x * blockDim.x) + threadIdx.x;
  output[thread_index] = input1[thread_index] + input2[thread_index];
}

/// @brief The main function
///
/// @param [in] argc    The number of arguments passed into the program
/// @param [in] argv    The actual arguments passed into the program
///
/// @return    EXIT_SUCCESS upon success
int main(int argc, const char* argv[])
{
  // 1. Check the number of arguments.
  if (argc != 3)
  {
    printf("ERROR: Incorrect number of arguments provided!\n");
    printf("----------------------------------------------\n");
    printf("Usage:\n");
    printf("  module_3_jrenga2.exe <number of blocks> <number of thread per block>\n");
    printf("\nPlease check your inputs and try again. Thank you! :)\n");

    return EXIT_FAILURE;
  }

  // 2. Attempt to retrieve the integer values of the parameters
  //    (a value less than or equal to 0 is considered invalid)
  int numBlocks = atoi(argv[1]);
  if (numBlocks <= 0)
  {
    printf("ERROR: Invalid number of blocks provided!\n");
    printf("-----------------------------------------\n");
    printf("The number of blocks must be a positive integer.\n");
    printf("\nPlease check your inputs and try again. Thank you! :)\n");

    return EXIT_FAILURE;
  }

  int numThreads = atoi(argv[2]);
  if (numThreads <= 0)
  {
    printf("ERROR: Invalid number of threads provided!\n");
    printf("------------------------------------------\n");
    printf("The number of threads per block must be a positive integer.\n");
    printf("\nPlease check your inputs and try again. Thank you! :)\n");

    return EXIT_FAILURE;
  }

  // 2.5 Check to see if the minimum number of threads has been achieved (64)
  if ((numBlocks * numThreads) < 64)
  {
    int threadCount = numBlocks * numThreads;
    printf("ERROR: Minimum number of threads not achieved!\n");
    printf("----------------------------------------------\n");
    printf("The total number of threads must be greater than 64. You have provided %d threads.\n", threadCount);
    printf("\nPlease check your inputs and try again. Thank you! :)\n");

    return EXIT_FAILURE;
  }
  // 3. Compute necessary array structures

  int array_size = numBlocks * numThreads;
  int array_size_in_bytes = array_size * sizeof(unsigned int);

  unsigned int* cpu_input_array1 = (unsigned int*) calloc(array_size, sizeof(unsigned int));
  unsigned int* cpu_input_array2 = (unsigned int*) calloc(array_size, sizeof(unsigned int));
  unsigned int* cpu_output_array = (unsigned int*) calloc(array_size, sizeof(unsigned int));

  int* gpu_input_1;
  int* gpu_input_2;
  int* gpu_output;

  int loopCounter = 0;

  for ( ; loopCounter < array_size; loopCounter++)
  {
    cpu_input_array1[loopCounter] = loopCounter;
    cpu_input_array2[loopCounter] = loopCounter * 2;
  }

  // 4. Prepare the GPU call

  hipMalloc((void **)&gpu_input_1, array_size_in_bytes);
  hipMemcpy(gpu_input_1, cpu_input_array1, array_size_in_bytes, hipMemcpyHostToDevice);

  hipMalloc((void **)&gpu_input_2, array_size_in_bytes);
  hipMemcpy(gpu_input_2, cpu_input_array2, array_size_in_bytes, hipMemcpyHostToDevice);

  hipMalloc((void **)&gpu_output,  array_size_in_bytes);
  hipMemcpy(gpu_output, cpu_output_array,  array_size_in_bytes, hipMemcpyHostToDevice);

  // 5. Perform the GPU call
  integer_addition<<< numBlocks, numThreads >>>(gpu_input_1, gpu_input_2, gpu_output);

  // 6. Retrieve the data
  hipMemcpy( cpu_output_array, gpu_output, array_size_in_bytes, hipMemcpyDeviceToHost);

  // 6.1 Free the remaining data
  hipFree(gpu_output);
  hipFree(gpu_input_2);
  hipFree(gpu_input_1);

  // 7. Display the data
  for ( loopCounter = 0; loopCounter < array_size; loopCounter++)
  {
    printf("Computed value (index %d): %u\n", loopCounter, cpu_output_array[loopCounter]);
  }

  free(cpu_output_array);
  free(cpu_input_array2);
  free(cpu_input_array1);

  return EXIT_SUCCESS;
}
