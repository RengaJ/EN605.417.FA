// Assignment For Module 04:
//
//
// Author: Justin Renga

// NOTE: This implementation was inspired from the sample code from
// https://devblogs.nvidia.com/how-optimize-data-transfers-cuda-cc/
// and, as such, there may be similarities between them.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCKSIZE 8
#define THREADCOUNT 256

// Define the array size as a constant value
#define ARRAY_SIZE 2048
#define BYTES ARRAY_SIZE * sizeof(unsigned int)

/// @brief The kernel function that will be used to test the functionality
///        of pinned and pageable memory.

__global__ void manipulateInput(unsigned int* input, unsigned int* output)
{
  const unsigned int thread_index = (blockIdx.x * blockDim.x) + threadIdx.x;
  output[thread_index] = (2 * input[thread_index]) - (thread_index/2);
}

/// @brief A function to test the usage of Pinned Memory.
float test_pinnedMemory()
{
  // Define data containers
  unsigned int* pinnedMemory;
  unsigned int* pinnedOutput;
  unsigned int* deviceIn;
  unsigned int* deviceOut;
  unsigned int* hostPageable;
  
  // Establish CUDA start/stop events
  hipEvent_t startEvent;
  hipEvent_t stopEvent;
  
  // Define the time container
  float time;
  
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);
  
  // Start the event timer:
  hipEventRecord(startEvent, 0);

  // Allocate Pageable Memory (on host)
  hostPageable = (unsigned int*) malloc(BYTES);

  // Allocate Pinned Memory (on host)
  // Need to use cudaMallocHost() to establish pinned memory
  hipHostMalloc((void**)&pinnedMemory, BYTES, hipHostMallocDefault);
  hipHostMalloc((void**)&pinnedOutput, BYTES, hipHostMallocDefault);
  hipMalloc((void**)&deviceIn, BYTES);
  hipMalloc((void**)&deviceOut, BYTES);

  // Initialize the pageable memory
  for (unsigned int i = 0; i < ARRAY_SIZE; ++i)
  {
    hostPageable[i] = (i + 1);
  }

  // Copy the pageable memory to the pinned memory
  memcpy(pinnedMemory, hostPageable, BYTES);
  memset(pinnedOutput, 0,            BYTES);
  
  // Copy the pinned memory to the device
  hipMemcpy(deviceIn,  pinnedMemory, BYTES, hipMemcpyHostToDevice);
  // Execute the KERNEL
  manipulateInput<<<BLOCKSIZE, THREADCOUNT>>>(deviceIn, deviceOut);
  // Obtain the output
  hipMemcpy(pinnedOutput, deviceOut, BYTES, hipMemcpyDeviceToHost);

  printf("Final Output Value:\n");
  printf("pinnedOutput[%d] = %u\n", ARRAY_SIZE-1, pinnedOutput[ARRAY_SIZE-1]);

  // Free Memory
  hipFree(deviceIn);
  hipFree(deviceOut);
  hipHostFree(pinnedOutput);
  hipHostFree(pinnedMemory);
  free(hostPageable);
  
  // Stop the event timer
  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  
  hipEventElapsedTime(&time, startEvent, stopEvent);
  
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);
  
  return time;
}

/// @ brief A function to test the usage of Pageable Memory
float test_pageableMemory()
{
  // Define data containers
  unsigned int* deviceIn;
  unsigned int* deviceOut;
  unsigned int* pageableInput;
  unsigned int* pageableOutput;
  
  // Establish CUDA start/stop events
  hipEvent_t startEvent;
  hipEvent_t stopEvent;
  
  // Define the time container
  float time;
  
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);
  
  // Start the event timer:
  hipEventRecord(startEvent, 0);
  
  // Begin the memory allocations
  
  // Allocate Pageable Memory (on host)
  pageableInput  = (unsigned int*) malloc(BYTES);
  pageableOutput = (unsigned int*) malloc(BYTES);
  hipMalloc((void**)&deviceIn, BYTES);
  hipMalloc((void**)&deviceOut, BYTES);
  
  // Initialize the pageable memory
  for (unsigned int i = 0; i < ARRAY_SIZE; ++i)
  {
    pageableInput[i]  = (i + 1);
    pageableOutput[i] = 0;
  }
  
  // Copy the pinned memory to the device
  hipMemcpy(deviceIn,  pageableInput, BYTES, hipMemcpyHostToDevice);
  hipMemcpy(deviceOut, pageableOutput, BYTES, hipMemcpyHostToDevice);

  // Execute the KERNEL
  manipulateInput<<<BLOCKSIZE, THREADCOUNT>>>(deviceIn, deviceOut);
  
  // Obtain the output
  hipMemcpy(pageableOutput, deviceOut, BYTES, hipMemcpyDeviceToHost);

  printf("Final Output Value:\n");
  printf("pageableOutput[%d] = %u\n", ARRAY_SIZE-1, pageableOutput[ARRAY_SIZE-1]);
  
  // Free Memory
  hipFree(deviceIn);
  hipFree(deviceOut);
  free(pageableInput);
  free(pageableOutput);

  // Stop the event timer
  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  
  hipEventElapsedTime(&time, startEvent, stopEvent);
  
  hipEventDestroy(startEvent);
  hipEventDestroy(stopEvent);
  
  return time;
}

/// @brief The main function
///
/// @param [in] argc    The number of arguments passed into the program
/// @param [in] argv    The actual arguments passed into the program
///
/// @return    EXIT_SUCCESS upon success
int main(int argc, const char* argv[])
{
  // Create an array that will hold the durations of the tests
  float duration[2];

  // Execute and store the results of the two tests
  duration[0] = test_pinnedMemory();
  duration[1] = test_pinnedMemory();

  // Output the results of the tests
  printf("Results:\n");
  printf("--------\n");
  printf("Pinned Memory   Test Duration: %3f seconds\n", duration[0]);
  printf("Pageable Memory Test Duration: %3f seconds\n", duration[1]);
 
  return EXIT_SUCCESS;
}
