#include "hip/hip_runtime.h"
// Introduction to GPU Programming
// Module 05: Shared and Constant Memory
// Author   : Justin Renga
// Date     : 03/03/2017

// Bibliography (source-list):
// [1] http://cuda-programming.blogspot.com/2013/01/what-is-constant-memory-in-cuda.html
// [2] constant_memory.cu
// [3] shared_memory2.cu

// #includes
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

#define ELEMENT_COUNT 2048

// Declare CONSTANT Memory here:
__constant__ static unsigned int MASKS[2];

// Declare static SHARED Memory here:
__shared__   unsigned int sharedMemory [ELEMENT_COUNT];

// Declare global host data here:
unsigned int initializedRNG;

// ---------------------------------------- DEVICE OPERATIONS -----------------------------------------

// @brief Perform a masking operation based on the evenness of the data
//
// @param [ in] data   An array that contains input data
// @param [out] output An array that contains the resulting data
__global__ void maskBits(const unsigned int*       data,
                               unsigned int* const output)
{
  // Compute the thread index
  const unsigned int threadIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  const unsigned int dataValue = data[threadIndex];
    
  // Apply the appropriate mask value and assign to the output
  output[threadIndex] = dataValue ^ MASKS[dataValue % 2];
}

// @brief Performs an array reversal
//
// @param [   in] The shared memory
// @param [inout] The device memory
// @param [   in] The number of elements
// @param [   in] The current thread index
__device__ void rotateArray(unsigned int* const data,
                            unsigned int* const deviceData,
                            unsigned int        elementCount,
                            unsigned int        threadIndex)
{
  // Get the next appropriately rotated thread index
  unsigned int rotatedIndex = elementCount - threadIndex - 1;
  
  data[threadIndex] = deviceData[rotatedIndex];
  
  __syncthreads();
  
  // Store the newly rotated memory back into global memory
  deviceData[threadIndex] = data[threadIndex];
}

// @brief A kernel function used to flip the contents of an array
__global__ void swapArray(unsigned int* data, const unsigned int elementCount)
{
  // Get the current thread index
  unsigned int threadIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  rotateArray(sharedMemory, data, elementCount, threadIndex);
}

// ----------------------------------------- HOST OPERATIONS -----------------------------------------

// @brief Initialize the Random number generator and ensure it only initializes one time
void initializeRandomNumbers()
{
  if (initializedRNG == 0)
  {
    srand(time(NULL));
    initializedRNG = 1;
  }
}

// @brief Generates a series of random numbers for the provided array, given the number of desired numbers
//        and the maximum (exclusive) value.
//
// @param [inout] data         The data array that will contain the random numbers
// @param [   in] elementCount The number of elements to store in the data array
// @param [   in] max          The maximum random number to use (exclusive)
void generateRandomNumbers(      unsigned int* data,
                           const unsigned int  elementCount,
                           const unsigned int  max)
{
  // Check to make sure the RNG has been initialized
  if (initializedRNG == 0)
  {
    // If not, initialize the RNG
    initializeRandomNumbers();
  }
  for (unsigned int i = 0; i < elementCount; ++i)
  {
    data[i] = rand() % max;
  }
}

// @brief Host function to run the GPU algorithms
//
// @param [in] threadCount Number of threads per block
// @param [in] blockCount  Number of blocks
// @param [in] maskMode    Should the algorithm to run be the mask algorithm?
void runAlgorithm(unsigned int threadCount, unsigned int blockCount, bool maskMode)
{
  static unsigned int elementCount = threadCount * blockCount;
  
  // Declare the device array size
  static unsigned int deviceSize = sizeof(unsigned int) * elementCount;
  
  // Allocate host memory to copy to/from the GPU
  unsigned int* hostInput  = (unsigned int*)malloc(deviceSize);
  unsigned int* hostOutput = (unsigned int*)calloc(elementCount, sizeof(unsigned int));
  
  // Populate the host memory with random numbers from 0 to elementCount - 1
  generateRandomNumbers(hostInput, elementCount, elementCount);
  
  // Allocate global memory for the GPU
  unsigned int* deviceInput;
  unsigned int* deviceOutput;
  
  hipMalloc((void**)&deviceInput , deviceSize);
  hipMalloc((void**)&deviceOutput, deviceSize);
  
  // Initialize the device output memory to zero
  hipMemset(deviceOutput, 0, deviceSize);
  
  // Copy the host input memory to GPU input memory
  hipMemcpy(deviceInput,  hostInput,  deviceSize, hipMemcpyHostToDevice);
  
    if (maskMode)
  {
    // Define the constant values for the MASK
    static unsigned int masks[2];
    
    // Mask ZERO: 00000001 00100011 01000101 01100111
    // Mask ONE : 10111111 11101010 11001101 00010110
    masks[0] = 0x01234567;
    masks[1] = 0xBFEACD16;
    
    // Copy the masks symbol from host to constant GPU memory:
    hipMemcpyToSymbol(HIP_SYMBOL(MASKS), masks, sizeof(unsigned int) * 2);
    
      // Execute the kernel
    maskBits<<<blockCount, threadCount>>>(deviceInput, deviceOutput);
  }
  else
  {
    swapArray<<<blockCount, threadCount>>>(deviceInput, elementCount);
  
    // Make sure the GPU has completed before copying everything over
    hipDeviceSynchronize();
  }
  
  if (maskMode)
  {
    // Copy memory from device to host for output
    hipMemcpy(hostOutput, deviceOutput, deviceSize, hipMemcpyDeviceToHost);
  }
  else
  {
    // Copy memory from device to host for output
    hipMemcpy(hostOutput, deviceInput, deviceSize, hipMemcpyDeviceToHost);
  }
  
  // Print the trimmed results (the first and last 4 values)
  for (unsigned int i = 0; i < 4; ++i)
  {
    printf("[INDEX %5u] Input: %4u   Output: %u\n", i, hostInput[i], hostOutput[i]);
  }
  
  printf("\n...Trimmed output...\n\n");
  
  for (unsigned int i = elementCount - 5; i < elementCount; ++i)
  {
    printf("[INDEX %5u] Input: %4u   Output: %u\n", i, hostInput[i], hostOutput[i]);
  }
  
  // Free the device memory
  hipFree(deviceInput );
  hipFree(deviceOutput);
  
  // Free the host memory
  free(hostInput);
  free(hostOutput);
  
  hostInput = NULL;
  hostOutput = NULL;
}

// @brief Display the proper program usage
void showUsage()
{
  printf("Invalid arguments provided. Please see the usage below:\n");
  printf("  module_3_jrenga2.exe <bc> <tpb>\n");
  printf("  bc  - The number of blocks to run with. Must be a positive integer.\n");
  printf("  tpb - The number of threads per blocks. Must be a positive integer.\n");
  printf("NOTE: The total number of threads (bc * tpb) must be greater than 64 \n");
  printf("      and less than 2048.\n");
  printf(" ** TERMINATING **\n");
}

// @brief Main Entry-Point
int main(int argc, char* argv[])
{
 // 1. Check the number of arguments.
  if (argc != 3)
  {
    showUsage();
    return EXIT_FAILURE;
  }

  // 2. Attempt to retrieve the integer values of the parameters
  //    (a value less than or equal to 0 is considered invalid)
  int numBlocks = atoi(argv[1]);
  if (numBlocks <= 0)
  {
    showUsage();
    return EXIT_FAILURE;
  }

  int numThreads = atoi(argv[2]);
  if (numThreads <= 0)
  {
    showUsage();
    return EXIT_FAILURE;
  }
  
  int totalThreads = numBlocks * numThreads;

  // 2.5 Check to see if the minimum number of threads has been achieved (64)
  if (totalThreads < 64 || totalThreads > 2048)
  {
    showUsage();
    return EXIT_FAILURE;
  }
  
  // Do some pre-processing to set up the random number generation
  initializedRNG = false;

  initializeRandomNumbers();
  
  printf("Swapping Algorithm (SHARED MEMORY) [BLOCKS = %2d, THREADS/BLOCK = %2d]\n", numBlocks, numThreads);
  printf("********************************************************************\n");
  
  // Run the swapping algorithm using shared memory
  runAlgorithm(numBlocks, numThreads, false);
  
  printf("Masking Algorithm (CONSTANT MEMORY) [BLOCKS = %2d, THREADS/BLOCK = %2d]\n", numBlocks, numThreads);
  printf("*********************************************************************\n");
  
  // Run the masking algorithm using constant memory
  runAlgorithm(numBlocks, numThreads, true);

  return EXIT_SUCCESS;
}
