#include "hip/hip_runtime.h"
/// Assignment 06: Local Register Memory
///
/// Author: Justin Renga

/// Two Kernels -- Same Operation
///
/// Operation: Take an integer (randomly generated) from two input arrays,
///            take their modulo (input1 % input2) and store the result.
///
/// Kernel 1: Use the global memory to perform the operation (using local memory as an
///           intermediate.

/// Kernel 2: Transfer the data from global memory to local memory, perform the operation,
///           then transfer back to global memory

// Bibliography (source-list):
// [1] register.cu
// [2] http://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__EVENT.html
// [3] https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/

#include <time.h>
#include <stdio.h>
#include <stdlib.h>

// Declare constant definitions here
#define UINT_SIZE sizeof(unsigned int)
#define INITIAL_DATA 0
#define THREAD_MIN 64
#define THREAD_MAX 4096

// Declare device constant memory here
__constant__ static unsigned int ADDITIVE_VALUES[16];

// Declare global host data here:
unsigned int initializedRNG;

// ---------------------------------------- DEVICE OPERATIONS -----------------------------------------

/// @brief GPU Kernel that utilizes only global data to perform a simple modular division operation. To
///        be used on conjunction with localModularDivide for comparison metrics. Algorithm performed:
///        output = input1 % input2
///
/// @param [ in] input1    The first of the two input arrays to be used in the modular division operation
/// @param [ in] input2    The second of the two input arrays to be used in the modular division operation
/// @param [out] output    The array containing the results of the modular division operation
__global__ void globalModularDivide(const unsigned int* const input1,
                                    const unsigned int* const input2,
                                          unsigned int* const output)
{
  // Compute the current thread index
  unsigned int thread_index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  // Perform the modular operation and store in the output array (without using local memory)
  output[thread_index] = input1[thread_index] % input2[thread_index];
}

/// @brief GPU Kernel that offloads the computations from global data completely, then re-inserts
///        the data back into global memory. To be used with globalModularDivide for comparison metrics.
///        Algorithm performed: output = input1 % input2
///
/// @param [ in] input1    The first of the two input arrays to be used in the modular division operation
/// @param [ in] input2    The second of the two input arrays to be used in the modular division operation
/// @param [out] output    The array containing the results of the modular division operation
__global__ void localModularDivide(const unsigned int* const input1,
                              const unsigned int* const input2,
                                    unsigned int* const output)
{
  // Compute the current thread index
  unsigned int thread_index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  // Create local registers to store the intermediate data for the algorithm
  unsigned int input1_local = input1[thread_index];
  unsigned int input2_local = input2[thread_index];
  
  // Create a local register that will store the result of the algorithm
  unsigned int output_local = input1_local % input2_local;
  
  // Store the result of the algorithm into the global array
  output[thread_index] = output_local;
}

__global__ void add_values_shared(unsigned int* deviceData, const unsigned int elementCount)
{
  // Declare externally defined shared memory
  __shared__ unsigned int sharedMemory[THREAD_MAX];
  
  // Compute the current thread index
  unsigned int threadIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  // Copy data from the device to the shared memory pool (and perform an operation using constant memory)
  sharedMemory[threadIndex] = deviceData[threadIndex];
  
  // Perform thread synchronization
  __syncthreads();
  
  unsigned int exponentPrimer = threadIndex % 2;
  
  for (unsigned int i = 0; i < 16; ++i)
  {
    unsigned int currentConstant =  ADDITIVE_VALUES[i];
    
    float value = powf(-1, exponentPrimer) * currentConstant;
    
    sharedMemory[threadIndex] += value;
  }
  
  __syncthreads();
  
  // Copy the data from the shared memory back to the device
  deviceData[threadIndex] = sharedMemory[elementCount - threadIndex - 1];
}

// ----------------------------------------- HOST OPERATIONS -----------------------------------------

// @brief Initialize the Random number generator and ensure it only initializes one time
__host__ void initializeRandomNumbers()
{
  if (initializedRNG == 0)
  {
    srand(time(NULL));
    initializedRNG = 1;
  }
}

// @brief Generates a series of random numbers for the provided array, given the number of desired numbers
//        and the maximum (exclusive) value.
//
// @param [inout] data         The data array that will contain the random numbers
// @param [   in] elementCount The number of elements to store in the data array
// @param [   in] max          The maximum random number to use (exclusive)
__host__ void generateRandomNumbers(      unsigned int* data,
                                    const unsigned int  elementCount,
                                    const unsigned int  max)
{
  // Check to make sure the RNG has been initialized
  if (initializedRNG == 0)
  {
    // If not, initialize the RNG
    initializeRandomNumbers();
  }
  
  // Generate random data between 0 and the provided maximum value
  for (unsigned int i = 0; i < elementCount; ++i)
  {
    data[i] = rand() % max;
  }
}

__host__ void run_gpu_algorithm(int blockCount, int threadCount)
{
  // Step 1: Compute the size of the device array based on the block and thread/per block counts
  unsigned int elementCount = threadCount * blockCount;
  unsigned int deviceSize   = UINT_SIZE   * elementCount;
  
  // Step 2: Allocate the necessary host memory (two input arrays and an output array
  //         (use malloc for the input, and calloc for the output since we want to modify
  //          the contents of the input PRIOR to executing the GPU kernels, but we want to
  //          initialize the output to 0 before copying the device output over)
  unsigned int* hostInput1 = (unsigned int*) malloc(deviceSize);
  unsigned int* hostInput2 = (unsigned int*) malloc(deviceSize);
  unsigned int* hostOutput = (unsigned int*) calloc(elementCount, UINT_SIZE);
  
  // Step 3: Populate the input arrays with random data, using the device size as the maximum value
  //         (the device size is used purely as a convenience number, and to ensure that the maximum
  //          value has the potential to change between each run)
  generateRandomNumbers(hostInput1, elementCount, deviceSize);
  generateRandomNumbers(hostInput2, elementCount, deviceSize);
  
  // Step 4: Allocate the GPU memory arrays
  unsigned int* deviceInput1;
  unsigned int* deviceInput2;
  unsigned int* deviceOutput;
  
  hipMalloc((void**)&deviceInput1, deviceSize);
  hipMalloc((void**)&deviceInput2, deviceSize);
  hipMalloc((void**)&deviceOutput, deviceSize);
  
  // Step 5: Populate the GPU input with the host input data
  hipMemcpy(deviceInput1, hostInput1, deviceSize, hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, deviceSize, hipMemcpyHostToDevice);
  
  // Step 6: Set the GPU output with all zeros
  hipMemset(deviceOutput, INITIAL_DATA, deviceSize);
  
  // Step 7: Initialize the CUDA event start/stop timers for benchmarking
  hipEvent_t stopLocalEvent;
  hipEvent_t stopGlobalEvent;
  hipEvent_t stopSharedEvent;
  hipEvent_t startLocalEvent;
  hipEvent_t startGlobalEvent;
  hipEvent_t startSharedEvent;

  hipEventCreate(&stopLocalEvent);  
  hipEventCreate(&stopGlobalEvent);
  hipEventCreate(&stopSharedEvent);
  hipEventCreate(&startLocalEvent);  
  hipEventCreate(&startGlobalEvent);
  hipEventCreate(&startSharedEvent);
  
  // Step 8: Invoke the global algorithm kernel with recording enabled
  hipEventRecord(startGlobalEvent);
  globalModularDivide<<<blockCount, threadCount>>>(deviceInput1, deviceInput2, deviceOutput);
  hipEventRecord(stopGlobalEvent);
  
  hipDeviceSynchronize();
  hipGetLastError();
  
  // Step 9: Retrieve the output from the global algorithm kernel
  hipMemcpy(hostOutput, deviceOutput, deviceSize, hipMemcpyDeviceToHost);
  
  // Step 10: Obtain the ms duration for the global algorithm
  hipEventSynchronize(stopGlobalEvent);
  
  float globalTimeMS = 0.0f;
  hipEventElapsedTime(&globalTimeMS, startGlobalEvent, stopGlobalEvent);
  
  // Step 11: Invoke the local algorithm kernel with recording enabled
  hipEventRecord(startLocalEvent);
  localModularDivide<<<blockCount, threadCount>>>(deviceInput1, deviceInput2, deviceOutput);
  hipEventRecord(stopLocalEvent);
  
  hipDeviceSynchronize();
  hipGetLastError();
  
  // Step 12: Retrieve the output from the local algorithm kernel
  hipMemcpy(hostOutput, deviceOutput, deviceSize, hipMemcpyDeviceToHost);
  
  // Step 13: Obtain the ms duration for the local algorithm
  hipEventSynchronize(stopLocalEvent);
  
  float localTimeMS = 0.0f;
  hipEventElapsedTime(&localTimeMS, startLocalEvent, stopLocalEvent);
  
  // Step 14: Upload the constant memory values to the device:
  unsigned int* constantMemory = (unsigned int*) malloc(deviceSize);
  generateRandomNumbers(constantMemory, elementCount, deviceSize);
  
  hipMemcpyToSymbol(HIP_SYMBOL(ADDITIVE_VALUES), constantMemory, UINT_SIZE * 16);
  
  // Step 15: Invoke the shared algorithm kernel with recording enabled
  hipEventRecord(startSharedEvent);
  add_values_shared<<<blockCount, threadCount>>>(deviceOutput, elementCount);
  hipEventRecord(stopSharedEvent);
  
  hipDeviceSynchronize();
  hipGetLastError();
  
  // Step 16: Retrieve the output from the global algorithm kernel
  hipMemcpy(hostOutput, deviceOutput, deviceSize, hipMemcpyDeviceToHost);
  
  // Step 17: Obtain the ms duration for the global algorithm
  hipEventSynchronize(stopSharedEvent);
  
  float sharedTimeMS = 0.0f;
  hipEventElapsedTime(&sharedTimeMS, startSharedEvent, stopSharedEvent);

  // Step 18: Display the results of the two operations
  printf("Block Count: %d\t Threads Per Block: %d\t", blockCount, threadCount);
  printf("Global Duration: %2f ms\t", globalTimeMS);
  printf("Shared Duration: %2f ms\t", sharedTimeMS);
  printf("Local  Duration: %2f ms\n", localTimeMS );
  
  // Step 19: Free device memory:
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  
  // Step 20: Free host memory
  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  
  // Step 21: Free constant memory
  free(constantMemory);
}

/// @brief determine if the provided number is a power of two
///
/// @param [in] number   The number to validate
///
/// @return True if the provided number is a power of two, false otherwise
__host__ bool isPowerOfTwo(const int number)
{
  // Initialize a mask a 00000000 00000000 00000000 00000001 (on 32-bit machines)
  int mask = 0x1;
  
  // Iterate over each of the bits in the mask, left shifting by one to
  // iterate to the next power of two
  for (unsigned int i = 0; i < sizeof(int) * 8; ++i, mask = mask << 1)
  {
   
    // Compute the resulting masked value
    int maskedValue = number & mask;
    
    // If the computed value is non-zero and is not the provided number,
    // the provided number is not a power of two:
    //
    // For example, 3 would not be a power of two:
    //           3 = 00000000 00000000 00000000 00000011
    //        mask = 00000000 00000000 00000000 00000010
    // maskedValue = 00000000 00000000 00000000 00000010
    // makedValue is non-zero (2), but is also not provided number (2 != 3)
    if (maskedValue != 0 && maskedValue != number)
    {
      return false;
    }
    
    // If the maskedValue is the provided number, then we've confirmed that the
    // value is a power of two
    if (maskedValue == number)
    {
      return true;
    }
  }
  
  // Return false if we've exhausted all possible powers of two the computer can handle
  return false;
}

// @brief Display the proper program usage
__host__ void showUsage()
{
  printf("Invalid arguments provided. Please see the usage below:\n");
  printf("  module_6_jrenga2.exe <bc> <tpb>\n");
  printf("  bc  - The maximum number of blocks to run with. Must be a positive integer and a power of two.\n");
  printf("  tpb - The maximum number of threads per blocks. Must be a positive integer and a power of two.\n");
  printf("NOTE: The maximum number of threads (bc * tpb) must be greater than %d \n", THREAD_MIN);
  printf("      and less than %d.\n", THREAD_MAX);
  printf(" ** TERMINATING **\n");
}

// @brief Main Entry-Point
int main(int argc, char* argv[])
{
 // 1. Check the number of arguments.
  if (argc != 3)
  {
    showUsage();
    return EXIT_FAILURE;
  }

  // 2. Attempt to retrieve the integer values of the parameters
  //    (a value less than or equal to 0 is considered invalid)
  int numBlocks = atoi(argv[1]);
  if (numBlocks <= 0 || !isPowerOfTwo(numBlocks))
  {
    showUsage();
    return EXIT_FAILURE;
  }

  int numThreads = atoi(argv[2]);
  if (numThreads <= 0 || !isPowerOfTwo(numThreads))
  {
    showUsage();
    return EXIT_FAILURE;
  }
  
  int totalThreads = numBlocks * numThreads;

  // 2.5 Check to see if the minimum number of threads has been achieved (64)
  if (totalThreads < THREAD_MIN || totalThreads > THREAD_MAX)
  {
    showUsage();
    return EXIT_FAILURE;
  }
  
  // Do some pre-processing to set up the random number generation
  initializedRNG = false;
  
  // Initialize the random numbers
  initializeRandomNumbers();
  
  // Iterate from 1 -> numBlocks and 1 -> numThreads to perform metrics on numerous configurations
  for (unsigned int blockCount = 1; blockCount <= numBlocks; blockCount = blockCount << 1)
  {
    for (unsigned int threadCount = 1; threadCount <= numThreads; threadCount = threadCount << 1)
    {
      run_gpu_algorithm(blockCount, threadCount);
    }
  }
  
  return EXIT_SUCCESS;
}
