#include "hip/hip_runtime.h"
/// Assignment 07: Streams and Events
///
/// Author: Justin Renga

#include <time.h>
#include <stdio.h>
#include <stdlib.h>

// Declare constant definitions here
#define DOUBLE_SIZE sizeof(double)
#define THREAD_MIN 64
#define THREAD_MAX 4096

// Declare global host data here:
unsigned int initializedRNG;

// ---------------------------------------- DEVICE OPERATIONS -----------------------------------------

// @brief GPU Kernel that takes three vector components (x,y,z) and produces the reciprocal magnitude of
//         the provided vector.   DEVICE ONLY
//
// @param [ in] x    The x-component of the vector
// @param [ in] y    The y-component of the vector
// @param [ in] z    The z-component of the vector
//
// @return  The reciprocal of the vector's magnitude
__device__ double getRMagnitude(const double x, const double y, const double z)
{
  // Return 1/ (Square Root(x^2 + y^2 + z^2))
  return rsqrt(pow(x, 2.0) + pow(y, 2.0) + pow(z, 2.0));
}

// @brief GPU kernel that takes three vector component vectors (x, y and z) and replaces
//        the contents with the unit vector components
//
// @param [inout] xComponents   INPUT: original x-components of the vectors | OUTPUT: normalized x-component
// @param [inout] yComponents   INPUT: original y-components of the vectors | OUTPUT: normalized y-component
// @param [inout] zComponents   INPUT: original z-components of the vectors | OUTPUT: normalized z-component
__global__ void normalize(double* xComponents, double* yComponents, double* zComponents)
{
  const unsigned int threadIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  double rMagnitude = getRMagnitude(xComponents[threadIndex], yComponents[threadIndex], zComponents[threadIndex]);
  
  xComponents[threadIndex] = xComponents[threadIndex] * rMagnitude;
  yComponents[threadIndex] = yComponents[threadIndex] * rMagnitude;
  zComponents[threadIndex] = zComponents[threadIndex] * rMagnitude;
}

// ----------------------------------------- HOST OPERATIONS -----------------------------------------

// @brief Initialize the Random number generator and ensure it only initializes one time
__host__ void initializeRandomNumbers()
{
  if (initializedRNG == 0)
  {
    srand(time(NULL));
    initializedRNG = 1;
  }
}

// @brief Generates a series of random numbers for the provided array, given the number of desired numbers
//        and the maximum (exclusive) value.
//
// @param [inout] data         The data array that will contain the random numbers
// @param [   in] elementCount The number of elements to store in the data array
// @param [   in] max          The maximum random number to use (exclusive)
__host__ void generateRandomNumbers(            double* data,
                                    const unsigned int  elementCount,
                                    const unsigned int  max)
{
  // Check to make sure the RNG has been initialized
  if (initializedRNG == 0)
  {
    // If not, initialize the RNG
    initializeRandomNumbers();
  }
  
  double division = max / (double)RAND_MAX;
  
  // Generate random data between 0 and the provided maximum value
  for (unsigned int i = 0; i < elementCount; ++i)
  {
    data[i] = rand() * division;
  }
}

__host__ void run_gpu_algorithm(int blockCount, int threadCount)
{
  // Compute the size of the device array based on the block and thread/per block counts
  static unsigned int elementCount = THREAD_MAX;
  static unsigned int deviceSize   = DOUBLE_SIZE * elementCount;
  
  // Allocate the necessary host memory (thee input arrays, three output arrays)
  double* hostXInput;
  double* hostYInput;
  double* hostZInput;
  
  double* hostXOutput;
  double* hostYOutput;
  double* hostZOutput;
  
  hipHostMalloc((void**)&hostXInput, deviceSize, hipHostMallocDefault);
  hipHostMalloc((void**)&hostYInput, deviceSize, hipHostMallocDefault);
  hipHostMalloc((void**)&hostZInput, deviceSize, hipHostMallocDefault);
  
  hipHostMalloc((void**)&hostXOutput, deviceSize, hipHostMallocDefault);
  hipHostMalloc((void**)&hostYOutput, deviceSize, hipHostMallocDefault);
  hipHostMalloc((void**)&hostZOutput, deviceSize, hipHostMallocDefault);
  
  // Populate the input arrays with random data, using the device size as the maximum value
  // (the device size is used purely as a convenience number, and to ensure that the maximum
  // value has the potential to change between each run)
  generateRandomNumbers(hostXInput, elementCount, deviceSize);
  generateRandomNumbers(hostYInput, elementCount, deviceSize);
  generateRandomNumbers(hostZInput, elementCount, deviceSize);
  
  // Allocate the GPU memory arrays
  double* deviceIO_X;
  double* deviceIO_Y;
  double* deviceIO_Z;
  
  hipMalloc((void**)&deviceIO_X, deviceSize);
  hipMalloc((void**)&deviceIO_Y, deviceSize);
  hipMalloc((void**)&deviceIO_Z, deviceSize);
  
  // Initialize the CUDA event start/stop timers for benchmarking
  hipEvent_t stopStreamEvent;
  hipEvent_t stopStandardEvent;
  hipEvent_t startStreamEvent;
  hipEvent_t startStandardEvent;
  
  hipEventCreate(&stopStreamEvent);  
  hipEventCreate(&stopStandardEvent);
  hipEventCreate(&startStreamEvent);
  hipEventCreate(&startStandardEvent);
  
  // Create the data stream
  hipStream_t data_stream;
  hipStreamCreate(&data_stream);
  
  // Start recording the stream process (starting from the first copy)
  hipEventRecord(startStreamEvent);
  
  // Perform the asynchronous memory copy from the host to the device, utilizing the stream
  hipMemcpyAsync(deviceIO_X, hostXInput, deviceSize, hipMemcpyHostToDevice, data_stream);
  hipMemcpyAsync(deviceIO_Y, hostYInput, deviceSize, hipMemcpyHostToDevice, data_stream);
  hipMemcpyAsync(deviceIO_Z, hostZInput, deviceSize, hipMemcpyHostToDevice, data_stream);
  
  // Invoke the normalization routine
  normalize<<<blockCount, threadCount>>>(deviceIO_X, deviceIO_Y, deviceIO_Z);
  
  // Retrieve the memory from the device asynchronously
  hipMemcpyAsync(hostXOutput, deviceIO_X, deviceSize, hipMemcpyDeviceToHost, data_stream);
  hipMemcpyAsync(hostYOutput, deviceIO_Y, deviceSize, hipMemcpyDeviceToHost, data_stream);
  hipMemcpyAsync(hostZOutput, deviceIO_Z, deviceSize, hipMemcpyDeviceToHost, data_stream);
  
  hipStreamSynchronize(data_stream);
  
  // Record the stop time for the stream process
  hipEventRecord(stopStreamEvent, 0);
  hipEventSynchronize(stopStreamEvent);
  
  hipDeviceSynchronize();
  hipGetLastError();
  
  // Obtain the ms duration for the stream normalization kernel invocation
  float streamTimeMS = 0.0f;
  hipEventElapsedTime(&streamTimeMS, startStreamEvent, stopStreamEvent);
  
  // Perform the same steps above, but without the stream's involvement
  hipEventRecord(startStandardEvent);
  
  hipMemcpy(deviceIO_X, hostXInput, deviceSize, hipMemcpyHostToDevice);
  hipMemcpy(deviceIO_Y, hostYInput, deviceSize, hipMemcpyHostToDevice);
  hipMemcpy(deviceIO_Z, hostZInput, deviceSize, hipMemcpyHostToDevice);
  
  // Invoke the normalization routine
  normalize<<<blockCount, threadCount>>>(deviceIO_X, deviceIO_Y, deviceIO_Z);
  
  hipMemcpy(hostXOutput, deviceIO_X, deviceSize, hipMemcpyDeviceToHost);
  hipMemcpy(hostYOutput, deviceIO_Y, deviceSize, hipMemcpyDeviceToHost);
  hipMemcpy(hostZOutput, deviceIO_Z, deviceSize, hipMemcpyDeviceToHost);
  
  hipEventRecord(stopStandardEvent);
  
  hipDeviceSynchronize();
  hipGetLastError();
  
  // Obtain the ms duration for the synchronous normalization kernel invocation
  hipEventSynchronize(stopStandardEvent);
  
  float standardTimeMS = 0.0f;
  hipEventElapsedTime(&standardTimeMS, startStandardEvent, stopStandardEvent);

  // Display the results of the two operations
  printf("Block Count: %d\t Threads Per Block: %d\t", blockCount, threadCount);
  printf("Stream Duration: %2f ms\t", streamTimeMS);
  printf("Standard Duration: %2f ms\n", standardTimeMS );
  
  // Free device memory:
  hipFree(deviceIO_X);
  hipFree(deviceIO_Y);
  hipFree(deviceIO_Z);
  
  // Free host memory
  hipHostFree(hostXInput);
  hipHostFree(hostYInput);
  hipHostFree(hostZInput);
  
  hipHostFree(hostXOutput);
  hipHostFree(hostYOutput);
  hipHostFree(hostZOutput);
}

/// @brief determine if the provided number is a power of two
///
/// @param [in] number   The number to validate
///
/// @return True if the provided number is a power of two, false otherwise
__host__ bool isPowerOfTwo(const int number)
{
  // Initialize a mask a 00000000 00000000 00000000 00000001 (on 32-bit machines)
  int mask = 0x1;
  
  // Iterate over each of the bits in the mask, left shifting by one to
  // iterate to the next power of two
  for (unsigned int i = 0; i < sizeof(int) * 8; ++i, mask = mask << 1)
  {
   
    // Compute the resulting masked value
    int maskedValue = number & mask;
    
    // If the computed value is non-zero and is not the provided number,
    // the provided number is not a power of two:
    //
    // For example, 3 would not be a power of two:
    //           3 = 00000000 00000000 00000000 00000011
    //        mask = 00000000 00000000 00000000 00000010
    // maskedValue = 00000000 00000000 00000000 00000010
    // makedValue is non-zero (2), but is also not provided number (2 != 3)
    if (maskedValue != 0 && maskedValue != number)
    {
      return false;
    }
    
    // If the maskedValue is the provided number, then we've confirmed that the
    // value is a power of two
    if (maskedValue == number)
    {
      return true;
    }
  }
  
  // Return false if we've exhausted all possible powers of two the computer can handle
  return false;
}

// @brief Display the proper program usage
__host__ void showUsage()
{
  printf("Invalid arguments provided. Please see the usage below:\n");
  printf("  module_7_jrenga2.exe <bc> <tpb>\n");
  printf("  bc  - The maximum number of blocks to run with. Must be a positive integer and a power of two.\n");
  printf("  tpb - The maximum number of threads per blocks. Must be a positive integer and a power of two.\n");
  printf("NOTE: The maximum number of threads (bc * tpb) must be greater than %d \n", THREAD_MIN);
  printf("      and less than %d.\n", THREAD_MAX);
  printf(" ** TERMINATING **\n");
}

// @brief Main Entry-Point
int main(int argc, char* argv[])
{
 // 1. Check the number of arguments.
  if (argc != 3)
  {
    showUsage();
    return EXIT_FAILURE;
  }

  // 2. Attempt to retrieve the integer values of the parameters
  //    (a value less than or equal to 0 is considered invalid)
  int numBlocks = atoi(argv[1]);
  if (numBlocks <= 0 || !isPowerOfTwo(numBlocks))
  {
    showUsage();
    return EXIT_FAILURE;
  }

  int numThreads = atoi(argv[2]);
  if (numThreads <= 0 || !isPowerOfTwo(numThreads))
  {
    showUsage();
    return EXIT_FAILURE;
  }
  
  int totalThreads = numBlocks * numThreads;

  // 2.5 Check to see if the minimum number of threads has been achieved (64)
  if (totalThreads < THREAD_MIN || totalThreads > THREAD_MAX)
  {
    showUsage();
    return EXIT_FAILURE;
  }
  
  // Do some pre-processing to set up the random number generation
  initializedRNG = false;
  
  // Initialize the random numbers
  initializeRandomNumbers();
  
  // Iterate from 1 -> numBlocks and 1 -> numThreads to perform metrics on numerous configurations
  for (unsigned int blockCount = 1; blockCount <= numBlocks; blockCount = blockCount << 1)
  {
    for (unsigned int threadCount = 1; threadCount <= numThreads; threadCount = threadCount << 1)
    {
      run_gpu_algorithm(blockCount, threadCount);
    }
  }
  
  return EXIT_SUCCESS;
}
