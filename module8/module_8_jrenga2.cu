#include "hip/hip_runtime.h"
/// Module 08 Assignment : NVidia GPU Accelerated Libraries Part 1
///
/// Author: Justin Renga
///
/// Date: March 21st, 2018
///
/// Purpose: This homework assignment is intended to show the usage of two
///          of the accelerated libraries showcased during this weeks lectures:
///            o cuFFT
///            o cuBLAS
///            o cuSPARSE
///            o cuSOLVER
///            o cuRAND
///
///          This implementation will utilize the cuRAND and cuSOLVER accelerated
///          libraries.

/// ------------------------------- FILE INCLUDES -------------------------------

// Include system files
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

// Include the cuFFT file
#include <hipfft/hipfft.h>

// Include the cuRAND files
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// Include the utility functions file (custom)
#include "module_8_jrenga2_utility.h"

/// ---------------------------- CONSTANT DEFINITIONS ---------------------------

#define THREAD_MIN 1
#define THREAD_MAX 4096

#define UINT_SIZE sizeof(unsigned int)
#define FLOAT_SIZE sizeof(float)
#define SAMPLE_SIZE 256
#define FLOAT2_SIZE sizeof(float2)

/// ---------------------------- GLOBAL DEFINITIONS -----------------------------
unsigned int initializedRNG;

/// ----------------------------- DEVICE OPERATIONS -----------------------------

__global__ void initializeCURAND(const unsigned int seed, hiprandState_t* stateList)
{
  // Initialize the cuRAND random-number generator on the device side.
  // Arguments into the hiprand_init function are as follows:
  //  1. The seed        - this is used to initialize pseudo-random generator
  //  2. The subsequence - the subsequence to start at
  //  3. The offset      - the number of values into the sequence that will be skipped upon
  //                       initialization
  //  4. The state       - The state pointer to initialize
  
  // In this case, each thread will have the same seed, but each core will have a different subsequence
  // of random numbers
  hiprand_init(seed, blockIdx.x, 0, &stateList[blockIdx.x]);
}

__global__ void generateRandom(unsigned int* result, hiprandState_t* stateList)
{
  unsigned int threadIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  unsigned int count = 0;
  float value = 0.0;
  
  hiprandState_t localState = stateList[blockIdx.x];
  
  for (unsigned int i = 0; i < SAMPLE_SIZE; ++i)
  {
    value = hiprand_uniform(&localState);
    
    if (value > 0.5)
    {
      ++count;
    }
  }
  
  stateList[blockIdx.x] = localState;
  result[threadIndex]   = count;
}

__global__ void addComplexNumbers(float2* complexA, float2* complexB)
{
  const unsigned int threadIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
  
  const unsigned int indexBase = 2 * threadIndex;
  
  complexA[indexBase].x = complexA[indexBase].x + complexB[indexBase].x;
  complexA[indexBase].y = complexA[indexBase].y + complexB[indexBase].y;
  
  complexA[indexBase + 1].x = complexA[indexBase + 1].x + complexB[indexBase + 1].x;
  complexA[indexBase + 1].y = complexA[indexBase + 1].y + complexB[indexBase + 1].y;
}

/// ------------------------------ HOST OPERATIONS ------------------------------

// @brief Initialize the Random number generator and ensure it only initializes one time
__host__ void initializeRandomNumbers()
{
  if (initializedRNG == 0)
  {
    srand(time(NULL));
    initializedRNG = 1;
  }
}

// @brief Generates a series of random numbers for the provided array, given the number of desired numbers
//        and the maximum (exclusive) value.
//
// @param [inout] data         The data array that will contain the random numbers
// @param [   in] elementCount The number of elements to store in the data array
// @param [   in] max          The maximum random number to use (exclusive)
__host__ void generateRandomNumbers(            float2* data,
                                    const unsigned int  elementCount,
                                    const unsigned int  max)
{
  // Check to make sure the RNG has been initialized
  if (initializedRNG == 0)
  {
    // If not, initialize the RNG
    initializeRandomNumbers();
  }
  
  float division = max / (float)RAND_MAX;
  
  // Generate random data between 0 and the provided maximum value
  for (unsigned int i = 0; i < elementCount; ++i)
  {
    data[i].x = rand() * division;
    
    // The imaginary component of the complex number will be set to zero
    data[i].y = 0;
  }
}

/// ============================== cuFFT OPERATIONS =============================
__host__ void executeFFT(const int blockCount, const int threadCount)
{
  // Obtain the element and device counts
  unsigned int elementCount = blockCount * threadCount * 2;
  unsigned int deviceSize   = elementCount * FLOAT2_SIZE;
  
  // Allocate the memory for the complex numbers (on the host)
  float2* hostComplex_A = (float2*) malloc(deviceSize);
  float2* hostComplex_B = (float2*) malloc(deviceSize);
  
  // Fill the complex numbers with data
  generateRandomNumbers(hostComplex_A, elementCount, deviceSize);
  generateRandomNumbers(hostComplex_B, elementCount, deviceSize);
  
  // Create the cuFFT complex data arrays (on the device)
  hipfftComplex* deviceComplex_A;
  hipfftComplex* deviceComplex_B;
  
  hipMalloc((void**)&deviceComplex_A, deviceSize);
  hipMalloc((void**)&deviceComplex_B, deviceSize);
  
  // Perform the memory copy from host to device
  hipMemcpy(deviceComplex_A, hostComplex_A, deviceSize, hipMemcpyHostToDevice);
  hipMemcpy(deviceComplex_B, hostComplex_B, deviceSize, hipMemcpyHostToDevice);
  
  // Create the handle to the cuFFT plan (and create the plan)
  // -- The plan to be created will be a 1-D plan with a
  //    Complex --> Complex transform type using single precision --
  hipfftHandle fftPlanHandle;
  hipfftPlan2d(&fftPlanHandle, blockCount, threadCount * 2, HIPFFT_C2C);
  
  // Create the timing events (one set for forward transform, one set for
  // inverse transform)
  hipEvent_t startForwardEvent;
  hipEvent_t startInverseEvent;
  hipEvent_t stopForwardEvent;
  hipEvent_t stopInverseEvent;
  
  hipEventCreate(&startForwardEvent);
  hipEventCreate(&startInverseEvent);
  hipEventCreate(&stopForwardEvent);
  hipEventCreate(&stopInverseEvent);
  
  // Perform the forward transformation (start the forward event recording)
  hipEventRecord(startForwardEvent);
  
  // Perform an in-place transform of both complex arrays (forward)
  hipfftExecC2C(fftPlanHandle, deviceComplex_A, deviceComplex_A, HIPFFT_FORWARD);
  hipfftExecC2C(fftPlanHandle, deviceComplex_B, deviceComplex_B, HIPFFT_FORWARD);
  
  // Record the end of the forward transform
  hipEventRecord(stopForwardEvent);
  hipEventSynchronize(stopForwardEvent);
  
  // Perform the custom kernel operation
  addComplexNumbers<<<blockCount, threadCount>>>(deviceComplex_A, deviceComplex_B);
  
  // Perform the inverse transformation (start the inverse event recording)
  hipEventRecord(startInverseEvent);
  
  // Perform the inverse transform of the first complex array
  hipfftExecC2C(fftPlanHandle, deviceComplex_A, deviceComplex_A, HIPFFT_BACKWARD);
  
  // Record the end of the inverse transform
  hipEventRecord(stopInverseEvent);
  hipEventSynchronize(stopInverseEvent);
 
  // Copy the contents of the resulting signal back to the host
  hipMemcpy(hostComplex_A, deviceComplex_A, deviceSize, hipMemcpyDeviceToHost);
  
    // Get the duration between the two events
  float cufftForwardDuration_MS;
  float cufftInverseDuration_MS;
  hipEventElapsedTime(&cufftForwardDuration_MS, startForwardEvent, stopForwardEvent);
  hipEventElapsedTime(&cufftInverseDuration_MS, startInverseEvent, stopInverseEvent);
  
  printf("\tForward Duration: %f ms\tInverse Duration: %f ms\n",
    cufftForwardDuration_MS, cufftInverseDuration_MS);
  
  // Destroy the cuFFT Plan
  hipfftDestroy(fftPlanHandle);

  // Free the device complex number memory
  hipFree(deviceComplex_A);
  hipFree(deviceComplex_B);
  
  // Free the host complex number memory
  free(hostComplex_A);
  free(hostComplex_B);
}  

/// ============================= cuRAND OPERATIONS =============================
__host__ void executeRandomNumbers(const int blockCount, const int threadCount)
{
  // Obtain the element and device counts
  unsigned int elementCount = blockCount * threadCount;
  unsigned int deviceSize   = elementCount * UINT_SIZE;
  
  // Allocate the host-side memory
  unsigned int* hostMemory = (unsigned int*) calloc(elementCount, UINT_SIZE);
  
  hiprandState_t* stateList;
  hipMalloc((void**)&stateList, elementCount * sizeof(hiprandState_t));
  
  // Allocate the CUDA memory
  unsigned int* deviceMemory;
  hipMalloc((void**)&deviceMemory, deviceSize);
  
  hipMemset(deviceMemory, 0, deviceSize);
  
  // Create events to be used for timing purposes
  hipEvent_t cuRandDevAPI_Start;
  hipEvent_t cuRandDevAPI_Stop;
  
  hipEventCreate(&cuRandDevAPI_Start);
  hipEventCreate(&cuRandDevAPI_Stop);
  
  // Record the start event
  hipEventRecord(cuRandDevAPI_Start);
  
  // Initialize cuRAND
  initializeCURAND<<<blockCount, 1>>>(time(NULL), stateList);
  
  // Invoke the RNG kernel
  generateRandom<<<blockCount, threadCount>>>(deviceMemory, stateList);

  // Perform the copy from the device to the host
  hipMemcpy(hostMemory, deviceMemory, deviceSize, hipMemcpyDeviceToHost);
  
  // Record the start event
  hipEventRecord(cuRandDevAPI_Stop);
  hipEventSynchronize(cuRandDevAPI_Stop);

  // Get the duration between the two events
  float cuRandDevAPI_DurationMS;
  hipEventElapsedTime(&cuRandDevAPI_DurationMS, cuRandDevAPI_Start, cuRandDevAPI_Stop);
  
  unsigned int finalResult = 0;
  for (unsigned int i = 0; i < elementCount; ++i)
  {
    finalResult += hostMemory[i];
  }
  
  float distribution = (finalResult / (float)(elementCount * SAMPLE_SIZE)) * 100.0f;
  
  // Display the results:
  printf("\tRandom Duration: %2f ms -- Distribution %03.2f%%\n", cuRandDevAPI_DurationMS, distribution);
  
  // Free the host memory
  free(hostMemory);
  
  // Free the device memory
  hipFree(deviceMemory);
  hipFree(stateList);
}


/// @brief Display the proper way to invoke the program.
__host__ void showProgramUsage()
{
  printf("Invalid arguments provided. Please see the usage below:\n");
  printf("  module_8_jrenga2.exe <bc> <tpb>\n");
  printf("  bc  - The maximum number of blocks to run with. Must be a positive integer and a power of two.\n");
  printf("  tpb - The maximum number of threads per blocks. Must be a positive integer and a power of two.\n");
  printf("NOTE: The maximum number of threads (bc * tpb) must be greater than %d \n", THREAD_MIN);
  printf("      and less than %d.\n", THREAD_MAX);
  printf(" ** TERMINATING **\n");
}

/// @brief The main function that will be used to drive the program.
///
/// @param [in] argc    The number of arguments passed into the program
/// @param [in] argv    The actual arguments passed into the program
///
/// @return An integer indicating the state of the program upon execution. If
///         successful, this will return EXIT_SUCCESS Otherwise, it will return
///         EXIT_FAILURE.
int main(int argc, char* argv[])
{
  // Check the number of arguments to determine how the program should operate
  if (argc != 3)
  {
    // If the number of arguments is not exactly 3, show the program usage and terminate
    // with a failure.
    showProgramUsage();
    
    return EXIT_FAILURE;
  }
  
  // Extract the maximum number of blocks to be used for the duration of the program.
  int maximumBlocks = atoi(argv[1]);
  
  // Extract the maximum number of threads to be used for each block for the duration of the program.
  int maximumThreads = atoi(argv[2]);
  
  // Ensure that the provided maximum number of blocks (and threads) are:
  //   1. Valid numbers (positive integer)
  //   2. A power of two
  if ((maximumBlocks  <= 0 || !isPowerOfTwo(maximumBlocks) ) ||
      (maximumThreads <= 0 || !isPowerOfTwo(maximumThreads))   )
  {
    // If any of the above conditions are violated for either the maximum number of blocks or
    // the maximum number of threads per block, show the program usage and terminate with a
    // failure.
    showProgramUsage();
    
    return EXIT_FAILURE;
  }
  
  // Compute the current total of threads
  int totalThreads = maximumBlocks * maximumThreads;
  
  // Ensure that the total number of threads is between the minimum and maximum thread counts
  // (defined above as THREAD_MIN and THREAD_MAX)
  if (totalThreads < THREAD_MIN || totalThreads > THREAD_MAX)
  {
    // If the total number of threads exceeds the bounds (in either direction), show the
    // program usage and terminate with a failure.
    showProgramUsage();
    
    return EXIT_FAILURE;
  }
  
  printf("Random Numbers:\n\n");
  
  // Iterate from 1 -> numBlocks and 1 -> numThreads to perform metrics on numerous configurations
  for (unsigned int blockCount = 1; blockCount <= maximumBlocks; blockCount = blockCount << 1)
  {
    for (unsigned int threadCount = 1; threadCount <= maximumThreads; threadCount = threadCount << 1)
    {
      printf("Blocks: %3d\tThreads: %3d", blockCount, threadCount);
      executeRandomNumbers(blockCount, threadCount);
    }
  }
  
  printf("FFT:\n\n");
  
  // Iterate from 1 -> numBlocks and 1 -> numThreads to perform metrics on numerous configurations
  for (unsigned int blockCount = 1; blockCount <= maximumBlocks; blockCount = blockCount << 1)
  {
    for (unsigned int threadCount = 1; threadCount <= maximumThreads; threadCount = threadCount << 1)
    {
      printf("Blocks: %3d\tThreads: %3d", blockCount, threadCount);
      executeFFT(blockCount, threadCount);
    }
  }
  
  // Reset the device to its original state
  hipDeviceReset();
  
  return EXIT_SUCCESS;
}