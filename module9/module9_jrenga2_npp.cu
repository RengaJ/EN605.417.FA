#include <time.h>
#include <string>
#include <sstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <npp.h>

// Declare global host data here:
unsigned int initializedRNG = 0;

// @brief Initialize the Random number generator and ensure it only initializes one time
void initializeRandomNumbers()
{
  if (initializedRNG == 0)
  {
    srand(time(NULL));
    initializedRNG = 1;
  }
}

/// @brief Generate random number data, perform a simple thresholding. NOTE: **VERY** INEFFICIENT
///
/// @param [in] lengthModifier  The modifier 
void threshold_random_data(unsigned int lengthModifier)
{
  // Compute the total size of the npp32u array as 32 left-shifted by lengthModifier
  int data_size = 32 << lengthModifier;
  
  // Allocate the space for the data array
  Npp32s* signal1_data = nppsMalloc_32s(data_size);
  
  NppStatus returnStatus;
  
  Npp32s randomValue;
  
  // Attempt to fill the array with data.
  // Note that this implementation is the only way that I was able
  // to figure out how to assign different data to the array, and
  // is VERY inefficient
  for (int i = 0; i < data_size; ++i)
  {
    randomValue = rand();
    
    returnStatus = nppsSet_32s(randomValue, signal1_data + i, 1);
    
    if (returnStatus != NPP_SUCCESS && returnStatus != NPP_NO_ERROR)
    {
      std::cout << "Something failed attempting to set signal 1 data" << std::endl;
      std::cout << "Failure Code: " << returnStatus << std::endl;
      return;
    }
    
  }
  
  // Create the device buffer
  Npp8u deviceBuffer = 0;
  
  // Get the event handles for recording
  hipEvent_t start;
  hipEvent_t stop;
  
  // Create the event handles
  hipEventCreate(&start);
  hipEventCreate(&stop);
  
  // Record the start
  hipEventRecord(start);
  
  // Get the maximum value of signal1_data
  Npp32s maximum_data_value;
  returnStatus = nppsMax_32s(signal1_data, data_size, &maximum_data_value, &deviceBuffer);
  
    if (returnStatus != NPP_SUCCESS && returnStatus != NPP_NO_ERROR)
    {
      std::cout << "Something failed attempting to get maximum" << std::endl;
      std::cout << "Failure Code: " << returnStatus << std::endl;
      return;
    }
  
  // Record the stop
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  
  // Get the time
  float duration = 0;
  hipEventElapsedTime(&duration, start, stop);
  
  // Print the results
  std::cout << "Data Size: " << data_size << "\t\tMaximum Value: " << maximum_data_value << "\tComputation Duration (ms): " << duration << std::endl;
  
  // Free the host data
  nppsFree((void*)signal1_data);
  
  hipDeviceReset();
}

// The main entry point into the program.
int main(int argc, char* argv[])
{
  initializeRandomNumbers();
  
  for (unsigned int i = 0; i < 20; ++i)
  {
    threshold_random_data(i);
  }
  return EXIT_SUCCESS;
}