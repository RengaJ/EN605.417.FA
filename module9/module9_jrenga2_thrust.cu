#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/generate.h>        // Allow for the 
#include <thrust/reduce.h>          // Include the reduce operation
#include <thrust/fill.h>            // Include the fill operation
#include <thrust/functional.h>      // Include the modulus operation
#include <thrust/adjacent_difference.h>

#include <time.h>
#include <iostream>

// http://docs.nvidia.com/cuda/thrust/index.html



// Perform a simple value replacement algorithm
void count_adj_diffs(const unsigned int offset)
{
  int data_size = 32 << offset;
  
  hipEvent_t adjDiffStart;
  hipEvent_t adjDiffStop;
  
  hipEventCreate(&adjDiffStart);
  hipEventCreate(&adjDiffStop);
  
  hipEventRecord(adjDiffStart);
  
  thrust::host_vector<unsigned int> host_data(data_size);
  thrust::device_vector<unsigned int> results(data_size);
  
  thrust::generate(host_data.begin(), host_data.end(), rand);
  thrust::device_vector<unsigned int> device_data(host_data.begin(), host_data.end());
  
  thrust::adjacent_difference(device_data.begin(), device_data.end(), results.begin());
  
  int result = thrust::reduce(results.begin(), results.end());
  
  hipEventRecord(adjDiffStop, 0);
  hipEventSynchronize(adjDiffStop);
  
  float adjDiffTime;
  
  hipEventElapsedTime(&adjDiffTime, adjDiffStart, adjDiffStop);
  
  std::cout << "Number of random numbers: " << data_size;
  if (data_size < 100000)
  {
    std::cout << "\t";
  }
  std::cout << "\tTotal difference size: " << result << "\tElapsed Time (ms): " << adjDiffTime << std::endl;
}

// Perfom a count of odd random numbers 32 << offset numbers
void add_odd_randoms(const unsigned int offset)
{
  int data_size = 32 << offset;
  
  hipEvent_t randomStart;
  hipEvent_t randomStop;
  
  hipEventCreate(&randomStart);
  hipEventCreate(&randomStop);
  
  hipEventRecord(randomStart);

  thrust::host_vector<unsigned int> host_data(data_size);
  
  thrust::generate(host_data.begin(), host_data.end(), rand);
  
  thrust::device_vector<unsigned int> device_data(host_data.begin(), host_data.end());
  thrust::device_vector<unsigned int> modulo_data(data_size);
  
  thrust::fill(modulo_data.begin(), modulo_data.end(), 2);
  
  thrust::transform(device_data.begin(), device_data.end(), modulo_data.begin(), device_data.begin(), thrust::modulus<unsigned int>());
  
  int count = thrust::reduce(device_data.begin(), device_data.end());
  
  hipEventRecord(randomStop, 0);
  hipEventSynchronize(randomStop);
  
  float randomTime;
  
  hipEventElapsedTime(&randomTime, randomStart, randomStop);
  
  std::cout << "Number of random numbers: " << data_size;
  if (data_size < 100000)
  {
    std::cout << "\t";
  }
  std::cout << "\tNumber of odd numbers: " << count << "\tElapsed Time (ms): " << randomTime << std::endl;
}


int main(int argc, char* argv[])
{
  srand(time(NULL));
  
  for (unsigned int i = 0; i < 20; ++i)
  {
    count_adj_diffs(i);
    add_odd_randoms(i);
  }

  return EXIT_SUCCESS;
}