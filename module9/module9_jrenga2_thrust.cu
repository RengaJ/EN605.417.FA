#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/generate.h>        // Allow for the 
#include <thrust/reduce.h>          // Include the reduce operation
#include <thrust/fill.h>            // Include the fill operation
#include <thrust/functional.h>      // Include the modulus operation
#include <thrust/adjacent_difference.h>

#include <time.h>
#include <iostream>

// Bibliography
// [1] http://docs.nvidia.com/cuda/thrust/index.html


// Adds the sum of the differences between random numbers
void count_adj_diffs(const unsigned int offset)
{
  // Compute the size of the data
  int data_size = 32 << offset;
  
  // Create events (for timing)
  hipEvent_t adjDiffStart;
  hipEvent_t adjDiffStop;
  
  hipEventCreate(&adjDiffStart);
  hipEventCreate(&adjDiffStop);
  
  // Record the current time
  hipEventRecord(adjDiffStart);
  
  // Create the host data and results vector (they are not data dependent)
  thrust::host_vector<unsigned int> host_data(data_size);
  thrust::device_vector<unsigned int> results(data_size);
  
  // Generate random data and place it in the host vector
  thrust::generate(host_data.begin(), host_data.end(), rand);
  
  // Create a device vector that copies the host data
  thrust::device_vector<unsigned int> device_data(host_data.begin(), host_data.end());
  
  // Take the adjacent differences between each element in the device vector
  thrust::adjacent_difference(device_data.begin(), device_data.end(), results.begin());
  
  // Add the differences together and obtain the result
  int result = thrust::reduce(results.begin(), results.end());
  
  // Stop recording
  hipEventRecord(adjDiffStop, 0);
  hipEventSynchronize(adjDiffStop);
  
  // Get the time duration
  float adjDiffTime;
  
  hipEventElapsedTime(&adjDiffTime, adjDiffStart, adjDiffStop);
  
  // Display the results
  std::cout << "Number of random numbers: " << data_size;
  
  // (perform some formatting)
  if (data_size < 100000)
  {
    std::cout << "\t";
  }
  std::cout << "\tTotal difference size: " << result << "\tElapsed Time (ms): " << adjDiffTime << std::endl;
}

// Perfom a count of odd random numbers 32 << offset numbers
void add_odd_randoms(const unsigned int offset)
{
  // Compute the size of the data
  int data_size = 32 << offset;
  
  // Create events (for timing)
  hipEvent_t randomStart;
  hipEvent_t randomStop;
  
  hipEventCreate(&randomStart);
  hipEventCreate(&randomStop);
  
  // Record the current time
  hipEventRecord(randomStart);

  // Create the host data vector
  thrust::host_vector<unsigned int> host_data(data_size);
  
  // Generate random data and place it in the host vector
  thrust::generate(host_data.begin(), host_data.end(), rand);
  
  // Create a device vector that copies the host data
  thrust::device_vector<unsigned int> device_data(host_data.begin(), host_data.end());
  
  // Create the vector that will contain the modulo value
  thrust::device_vector<unsigned int> modulo_data(data_size);
  
  // Fill the modulo vector with 2 (all data values will be modulo 2)
  thrust::fill(modulo_data.begin(), modulo_data.end(), 2);
  
  // Perform the modulo opereration in place
  thrust::transform(device_data.begin(), device_data.end(), modulo_data.begin(), device_data.begin(), thrust::modulus<unsigned int>());
  
  // Add the number of 1's together (the number of odd numbers)
  int count = thrust::reduce(device_data.begin(), device_data.end());
  
  // Stop recording
  hipEventRecord(randomStop, 0);
  hipEventSynchronize(randomStop);
  
  // Get the time duration
  float randomTime;
  
  hipEventElapsedTime(&randomTime, randomStart, randomStop);
  
  // Display the results
  std::cout << "Number of random numbers: " << data_size;
  
  // (perform some formatting)
  if (data_size < 100000)
  {
    std::cout << "\t";
  }
  std::cout << "\tNumber of odd numbers: " << count << "\tElapsed Time (ms): " << randomTime << std::endl;
}

// The main entry point (inputs are ignored)
int main(int argc, char* argv[])
{
  // Seed the random number generator with the current time
  srand(time(NULL));
  
  // Iterate 20 times, calling each algorithm
  for (unsigned int i = 0; i < 20; ++i)
  {
    count_adj_diffs(i);
    add_odd_randoms(i);
  }

  return EXIT_SUCCESS;
}